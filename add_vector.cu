#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Índice global
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 20; 
    size_t size = N * sizeof(float);

    // Aloca memória no host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Aloca memória na GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copia os dados do host para a GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define o número de threads e blocos
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copia o resultado de volta para o host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verifica o resultado
    for (int i = 0; i < 100; i++) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    // Libera a memória
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
